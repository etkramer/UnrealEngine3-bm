#include "hip/hip_runtime.h"
// Copyright NVIDIA Corporation 2007 -- Ignacio Castano <icastano@nvidia.com>
// 
// Permission is hereby granted, free of charge, to any person
// obtaining a copy of this software and associated documentation
// files (the "Software"), to deal in the Software without
// restriction, including without limitation the rights to use,
// copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the
// Software is furnished to do so, subject to the following
// conditions:
// 
// The above copyright notice and this permission notice shall be
// included in all copies or substantial portions of the Software.
// 
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
// EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES
// OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
// NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT
// HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY,
// WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR
// OTHER DEALINGS IN THE SOFTWARE.

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include "CudaMath.h"

#define THREAD_NUM 64		// Number of threads per block.

#if __DEVICE_EMULATION__
#define __debugsync() __syncthreads()
#else
#define __debugsync()
#endif

typedef unsigned short ushort;
typedef unsigned int uint;

template <class T> 
__device__ inline void swap(T & a, T & b)
{
	T tmp = a;
	a = b;
	b = tmp;
}

__constant__ float3 kColorMetric = { 1.0f, 1.0f, 1.0f };


////////////////////////////////////////////////////////////////////////////////
// Round color to RGB565 and expand
////////////////////////////////////////////////////////////////////////////////
inline __device__ float3 roundAndExpand(float3 v, ushort * w)
{
    v.x = rintf(__saturatef(v.x) * 31.0f);
    v.y = rintf(__saturatef(v.y) * 63.0f);
    v.z = rintf(__saturatef(v.z) * 31.0f);
    *w = ((ushort)v.x << 11) | ((ushort)v.y << 5) | (ushort)v.z;
    v.x *= 0.03227752766457f; // approximate integer bit expansion.
    v.y *= 0.01583151765563f;
    v.z *= 0.03227752766457f;
    return v;
}


////////////////////////////////////////////////////////////////////////////////
// Evaluate permutations
////////////////////////////////////////////////////////////////////////////////
static __device__ float evalPermutation4(const float3 * colors, uint permutation, ushort * start, ushort * end)
{
    // Compute endpoints using least squares.
    float alpha2_sum = 0.0f;
    float beta2_sum = 0.0f;
    float alphabeta_sum = 0.0f;
    float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);
    float3 betax_sum = make_float3(0.0f, 0.0f, 0.0f);

    // Compute alpha & beta for this permutation.
    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        float beta = (bits & 1);
        if (bits & 2) beta = (1 + beta) / 3.0f;
        float alpha = 1.0f - beta;
    
        alpha2_sum += alpha * alpha;
        beta2_sum += beta * beta;
        alphabeta_sum += alpha * beta;
        alphax_sum += alpha * colors[i];
        betax_sum += beta * colors[i];
    }

    // alpha2, beta2, alphabeta and factor could be precomputed for each permutation, but it's faster to recompute them.
    const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

    float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
    float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;
    
    // Round a, b to the closest 5-6-5 color and expand...
    a = roundAndExpand(a, start);
    b = roundAndExpand(b, end);

    // compute the error
    float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

    return dot(e, kColorMetric);
}


static __device__ float evalPermutation3(const float3 * colors, uint permutation, ushort * start, ushort * end)
{
    // Compute endpoints using least squares.
    float alpha2_sum = 0.0f;
    float beta2_sum = 0.0f;
    float alphabeta_sum = 0.0f;
    float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);
    float3 betax_sum = make_float3(0.0f, 0.0f, 0.0f);

    // Compute alpha & beta for this permutation.
    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        float beta = (bits & 1);
        if (bits & 2) beta = 0.5f;
        float alpha = 1.0f - beta;
    
        alpha2_sum += alpha * alpha;
        beta2_sum += beta * beta;
        alphabeta_sum += alpha * beta;
        alphax_sum += alpha * colors[i];
        betax_sum += beta * colors[i];
    }

    const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

    float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
    float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;
    
    // Round a, b to the closest 5-6-5 color and expand...
    a = roundAndExpand(a, start);
    b = roundAndExpand(b, end);

    // compute the error
    float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

    return dot(e, kColorMetric);
}


////////////////////////////////////////////////////////////////////////////////
// Sort colors
////////////////////////////////////////////////////////////////////////////////
__device__ void sortColors(float * values, float3 * colors, int * xrefs)
{
#if __DEVICE_EMULATION__

    if (threadIdx.x == 0) 
    {
        for( int i = 0; i < 16; ++i )
        {
			xrefs[i] = i;
		}
        
        // Use a sequential sort on emulation.
        for( int i = 0; i < 16; ++i )
        {
            for( int j = i; j > 0 && values[j] < values[j - 1]; --j )
            {
                swap( values[j], values[j - 1] );
                swap( xrefs[j], xrefs[j - 1] );
            //    swap( colors[j], colors[j - 1] );
            }
        }
        
        float3 tmp[16];
        for( int i = 0; i < 16; ++i ) 
        {
			tmp[i] = colors[i];
		}
        
        for( int i = 0; i < 16; ++i )
        {
            int xid = xrefs[i];
            colors[i] = tmp[xid];
        }
    }

#else
    int tid = threadIdx.x;

	xrefs[tid] = tid;

    // Parallel bitonic sort.
    for (int k = 2; k <= 16; k *= 2)
    {
        // bitonic merge:
        for (int j = k / 2; j>0; j /= 2)
        {
            int ixj = tid ^ j;
            
            if (ixj > tid) {
                // @@ Optimize these branches.
                if ((tid & k) == 0) {
                    if (values[xrefs[tid]] > values[xrefs[ixj]]) {
                    //    swap(values[tid], values[ixj]);
                        swap(colors[tid], colors[ixj]);
                        swap(xrefs[tid], xrefs[ixj]);
                    }
                }
                else {
                    if (values[xrefs[tid]] < values[xrefs[ixj]]) {
                    //    swap(values[tid], values[ixj]);
                        swap(colors[tid], colors[ixj]);
                        swap(xrefs[tid], xrefs[ixj]);
                    }
                }
            }
        }
    }
#endif

    // It would be faster to avoid color swaps during the sort, but there
    // are compiler bugs preventing that.
#if 0
	float3 tmp = colors[xrefs[tid]];
    colors[tid] = tmp;
#endif
}

// This sort is faster, but does not sort correctly elements with the same value.
__device__ void sortColors2(float * values, float3 * colors, int * cmp)
{
	int tid = threadIdx.x;

	cmp[tid] = (values[0] < values[tid]);
	cmp[tid] += (values[1] < values[tid]);
	cmp[tid] += (values[2] < values[tid]);
	cmp[tid] += (values[3] < values[tid]);
	cmp[tid] += (values[4] < values[tid]);
	cmp[tid] += (values[5] < values[tid]);
	cmp[tid] += (values[6] < values[tid]);
	cmp[tid] += (values[7] < values[tid]);
	cmp[tid] += (values[8] < values[tid]);
	cmp[tid] += (values[9] < values[tid]);
	cmp[tid] += (values[10] < values[tid]);
	cmp[tid] += (values[11] < values[tid]);
	cmp[tid] += (values[12] < values[tid]);
	cmp[tid] += (values[13] < values[tid]);
	cmp[tid] += (values[14] < values[tid]);
	cmp[tid] += (values[15] < values[tid]);
	
	float3 tmp = colors[tid];
	colors[cmp[tid]] = tmp;
}



////////////////////////////////////////////////////////////////////////////////
// Find index with minimum error
////////////////////////////////////////////////////////////////////////////////
__device__ void minimizeError(float * errors, int * indices)
{
	const int idx = threadIdx.x;

#if __DEVICE_EMULATION__

	for(int d = THREAD_NUM/2; d > 0; d >>= 1)
	{
		__syncthreads();

		if (idx < d)
		{
			float err0 = errors[idx];
			float err1 = errors[idx + d];
			
			if (err1 < err0) {
				errors[idx] = err1;
				indices[idx] = indices[idx + d];
			}
		}
	}

#else

	for(int d = THREAD_NUM/2; d > 32; d >>= 1)
	{
		__syncthreads();

		if (idx < d)
		{
			float err0 = errors[idx];
			float err1 = errors[idx + d];
			
			if (err1 < err0) {
				errors[idx] = err1;
				indices[idx] = indices[idx + d];
			}
		}
	}

	// unroll last 6 steps 
	if (idx <= 32)
	{
		if (errors[idx + 32] < errors[idx]) {
			errors[idx] = errors[idx + 32];
			indices[idx] = indices[idx + 32];
		}
		if (errors[idx + 16] < errors[idx]) {
			errors[idx] = errors[idx + 16];
			indices[idx] = indices[idx + 16];
		}
		if (errors[idx + 8] < errors[idx]) {
			errors[idx] = errors[idx + 8];
			indices[idx] = indices[idx + 8];
		}
		if (errors[idx + 4] < errors[idx]) {
			errors[idx] = errors[idx + 4];
			indices[idx] = indices[idx + 4];
		}
		if (errors[idx + 2] < errors[idx]) {
			errors[idx] = errors[idx + 2];
			indices[idx] = indices[idx + 2];
		}
		if (errors[idx + 1] < errors[idx]) {
			errors[idx] = errors[idx + 1];
			indices[idx] = indices[idx + 1];
		}
	}
#endif
}


////////////////////////////////////////////////////////////////////////////////
// Compress color block
////////////////////////////////////////////////////////////////////////////////
__global__ void compress(const uint * permutations, const uint * image, uint * result)
{
	const int bid = blockIdx.x;
	const int idx = threadIdx.x;
	
	__shared__ float3 colors[16];
	__shared__ float dps[16];
	__shared__ int xrefs[16];
	
	if (idx < 16)
	{
		// Read color.
		uint c = image[(bid) * 16 + idx];
	
		// No need to synchronize, 16 < warp size.
#if __DEVICE_EMULATION__
		} __debugsync(); if (idx < 16) {
#endif
		
		// Copy color to shared mem.
		colors[idx].z = ((c >> 0) & 0xFF) * (1.0f / 255.0f);
		colors[idx].y = ((c >> 8) & 0xFF) * (1.0f / 255.0f);
		colors[idx].x = ((c >> 16) & 0xFF) * (1.0f / 255.0f);
		
#if __DEVICE_EMULATION__
		} __debugsync(); if (idx < 16) {
#endif

		// Sort colors along the best fit line.
		float3 axis = bestFitLine(colors);
		
		dps[idx] = dot(colors[idx], axis);
		
#if __DEVICE_EMULATION__
		} __debugsync(); if (idx < 16) {
#endif
		
		sortColors(dps, colors, xrefs);
	}
	
	ushort bestStart, bestEnd;
	uint bestPermutation;
	float bestError = FLT_MAX;
	
	__syncthreads();
	
	for(int i = 0; i < 16; i++)
	{
		if (i == 15 && idx >= 32) break;
		
		ushort start, end;
		uint permutation = permutations[idx + THREAD_NUM * i];
		float error = evalPermutation4(colors, permutation, &start, &end);
		
		if (error < bestError)
		{
			bestError = error;
			bestPermutation = permutation;
			bestStart = start;
			bestEnd = end;
		}
	}

	if (bestStart < bestEnd)
	{
		swap(bestEnd, bestStart);
		bestPermutation ^= 0x55555555;	// Flip indices.
	}

	for(int i = 0; i < 3; i++)
	{
		if (i == 2 && idx >= 32) break;
		
		ushort start, end;
		uint permutation = permutations[idx + THREAD_NUM * i];
		float error = evalPermutation3(colors, permutation, &start, &end);
		
		if (error < bestError)
		{
			bestError = error;
			bestPermutation = permutation;
			bestStart = start;
			bestEnd = end;
			
			if (bestStart > bestEnd)
			{
				swap(bestEnd, bestStart);
				bestPermutation ^= (~bestPermutation >> 1) & 0x55555555;	// Flip indices.
			}
		}
	}
	
	if (bestStart == bestEnd)
	{
		bestPermutation = 0;
	}
	
	__syncthreads();
	
	// Use a parallel reduction to find minimum error.
	__shared__ float errors[THREAD_NUM];
	__shared__ int indices[THREAD_NUM];
	
	errors[idx] = bestError;
	indices[idx] = idx;
	
	minimizeError(errors, indices);
	
	__syncthreads();
	
	// Only write the result of the winner thread.
	if (idx == indices[0])
	{
		// Reorder permutation.
		uint perm = 0;
		for(int i = 0; i < 16; i++)
		{
			int ref = xrefs[i];
			perm |= ((bestPermutation >> (2 * i)) & 3) << (2 * ref);
		}
		
		// Write endpoints. (bestStart, bestEnd)
		result[2 * bid + 0] = (bestEnd << 16) | bestStart;
		
		// Write palette indices (permutation).
		result[2 * bid + 1] = perm;
	}
}


////////////////////////////////////////////////////////////////////////////////
// Launch kernel
////////////////////////////////////////////////////////////////////////////////
extern "C" void compressKernel(uint blockNum, uint * d_data, uint * d_result, uint * d_bitmaps, float weights[3])
{
	// Set constants.
	hipMemcpyToSymbol(HIP_SYMBOL(kColorMetric), weights, sizeof(float) * 3, 0);

	compress<<<blockNum, THREAD_NUM>>>(d_bitmaps, d_data, d_result);
}

